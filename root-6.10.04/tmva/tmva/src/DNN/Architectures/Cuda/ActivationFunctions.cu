// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the activation functions for the TCuda      //
 // implementation of the low-level interface.                   //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"

namespace TMVA
{
namespace DNN
{

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::IdentityDerivative(TCudaMatrix<AFloat> & B,
                                           const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::IdentityDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       (int) B.GetNrows(),
       (int) B.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Relu(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Relu<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::ReluDerivative(TCudaMatrix<AFloat> & B,
                                       const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::ReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Sigmoid(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Sigmoid<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::SigmoidDerivative(TCudaMatrix<AFloat> & B,
                                          const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SigmoidDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Tanh(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Tanh<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::TanhDerivative(TCudaMatrix<AFloat> & B,
                                       const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::TanhDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::SymmetricRelu(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SymmetricRelu<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::SymmetricReluDerivative(TCudaMatrix<AFloat> & B,
                                                const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SymmetricReluDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::SoftSign(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SoftSign<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::SoftSignDerivative(TCudaMatrix<AFloat> & B,
                                           const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::SoftSignDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Gauss(TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Gauss<<<gridDims, blockDims, 0, s>>>(
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
}

//______________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::GaussDerivative(TCudaMatrix<AFloat> & B,
                                    const TCudaMatrix<AFloat> & A)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(B);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::GaussDerivative<<<gridDims, blockDims, 0, s>>>(
       B.GetDataPointer(),
       A.GetDataPointer(),
       (int) A.GetNrows(),
       (int) A.GetNcols());
   B.SetComputeStream(s);
}

} // namespace DNN
} // namespace TMVA
